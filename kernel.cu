#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <omp.h>

// CUDA header
#include <hip/hip_runtime.h>

using namespace std;

struct RsaPublicKey {
    long long e, m;
};

struct RsaPrivateKey {
    long long d, m;
};

// Нахождение общего делителя
long long gcd(long long a, long long b) {
    if (b == 0)
        return a;
    else
        return gcd(b, a % b);
}

// Решение Диофантова уравнения a*x + b*y = 1
void DiophantEquation(long long a, long long b, long long& x, long long& y) {
    long long a11 = 1, a12 = 0, a21 = 0, a22 = 1;

    while (true) {
        long long r = a % b;
        if (r == 0) {
            x = a12;
            y = a22;
            return;
        }
        else {
            long long q = a / b;
            long long save12 = a12;
            long long save22 = a22;
            a12 = a11 - save12 * q;
            a22 = a21 - save22 * q;
            a11 = save12;
            a21 = save22;
            a = b;
            b = r;
        }
    }
}

// Поиск числа y, такого что (x*y)%m == 1
long long FindNumber(long long x, long long m) {
    long long y, sux;
    DiophantEquation(x, m, y, sux);

    while (y < 0)
        y += m;

    return y;
}

// Вычисление (a в степени b)%m
long long Power(long long a, long long b, long long m) {
    a %= m;
    long long res = a;
    for (long long i = 1; i < b; i++)
        res = (res * a) % m;

    return res;
}

// Генерация пары ключей
void GenerateKeyPair(long long p, long long q, long long e, RsaPublicKey& pub, RsaPrivateKey& pri) {
    if ((gcd(e, p - 1) != 1) || (gcd(e, q - 1) != 1)) {
        printf("GenerateKeyPair: Incorrect parameters\n");
        exit(1);
    }

    pub.m = p * q;
    pub.e = e;
    pri.m = p * q;

    // Функция Эйлера
    long long phi_m = (p - 1) * (q - 1);
    pri.d = FindNumber(e, phi_m);
}

// Сообщение должно быть < key. m
long long Encode(long long source, RsaPublicKey& key) {
    return Power(source, key.e, key.m);
}

long long Decode(long long source, RsaPrivateKey& key) {
    return Power(source, key.d, key.m);
}

// Ядро CUDA для параллельного кодирования
__global__ void parallelEncode(long long* arr, long long* result, long long e, long long m, int l) { //  функция-ядро CUDA, которая будет выполняться на GPU. global это значит
    /* индекс текущей потоковой области, вычисляемый на основе сетки блоков и размера блока.

    blockIdx.x представляет индекс блока, 
    blockDim.x - количество потоков в блоке,
    threadIdx.x - индекс потока внутри блока. 

    Это вычисление индекса помогает определить конкретный элемент массива, который будет обработан текущим потоком. */
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < l) {
        result[i] = 1; // initialize result
        for (long long j = 0; j < e; j++) {
            result[i] = (result[i] * arr[i]) % m;
        }
    }
}

// Ядро CUDA для параллельного декодирования
__global__ void parallelDecode(long long* arr, long long* result, long long d, long long m, int l) {

    // смотреть код выше
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < l) {
        result[i] = 1; // initialize result
        for (long long j = 0; j < d; j++) {
            result[i] = (result[i] * arr[i]) % m;
        }
    }
}

int main() {
    setlocale(LC_CTYPE, "Russian_Russia.1251");
    long long p = 29;
    long long q = 19;
    long long e = 47;
    long long arr[] = { 19, 13, 30, 350, 500, 19, 13, 30, 350, 500 };
    long long l = 10;

    RsaPublicKey pub_key;
    RsaPrivateKey pri_key;

    GenerateKeyPair(p, q, e, pub_key, pri_key);
    long long enc;
    long long dec = 0;
    long long i = 0;

    // Последовательная версия
    clock_t start_time = clock();
    for (i = 0; i < l; i++) {
        long long source = arr[i] % pub_key.m;
        enc = Encode(source, pub_key);
        dec = Decode(enc, pri_key);
        cout << "Последовательный: Оригинальный: " << source << " Зашифрованный: " << enc << " Расшифрованный: " << dec << endl;
    }

    clock_t end_time = clock();
    double sequential_time = double(end_time - start_time) / CLOCKS_PER_SEC;


    // Параллельная версия с использованием куды
    // размечаем память
    long long* d_arr;
    long long* d_enc_result;
    long long* d_dec_result;
    hipMalloc((void**)&d_arr, l * sizeof(long long));
    hipMalloc((void**)&d_enc_result, l * sizeof(long long));
    hipMalloc((void**)&d_dec_result, l * sizeof(long long));

    // копируем данные в девайс
    hipMemcpy(d_arr, arr, l * sizeof(long long), hipMemcpyHostToDevice);

    // устанавливаем сетку и размер блоков
    int block_size = 256;
    int grid_size = (l + block_size - 1) / block_size;

    // запускаем ядра куда
    start_time = clock();
    parallelEncode << <grid_size, block_size >> > (d_arr, d_enc_result, pub_key.e, pub_key.m, l);
    hipDeviceSynchronize(); // Wait for the GPU to finish


    parallelDecode << <grid_size, block_size >> > (d_enc_result, d_dec_result, pri_key.d, pri_key.m, l);

    // измерим скорость выполнения параллельной версии
    end_time = clock();
    double cuda_time = double(end_time - start_time) / CLOCKS_PER_SEC;

    // рассчитаем ускорение
    double speedup_cuda = sequential_time / cuda_time;

    // окончание измерения


    hipDeviceSynchronize(); // Wait for the GPU to finish

    // скопируем результат обратно в хост
    long long* h_enc_result = new long long[l];
    long long* h_dec_result = new long long[l];
    hipMemcpy(h_enc_result, d_enc_result, l * sizeof(long long), hipMemcpyDeviceToHost);
    hipMemcpy(h_dec_result, d_dec_result, l * sizeof(long long), hipMemcpyDeviceToHost);

    // вывод параллельных результатов
    for (i = 0; i < l; i++) {
        long long source = arr[i] % pub_key.m;
        cout << "Параллельный (CUDA): Оригинальный: " << source << " Зашифрованный: " << h_enc_result[i] << " Расшифрованный: "
            << h_dec_result[i] << endl;
    }


    cout << "Ускорение (CUDA): " << speedup_cuda << endl;
    // освобождаем
    hipFree(d_arr);
    hipFree(d_enc_result);
    hipFree(d_dec_result);

    // очищаем
    delete[] h_enc_result;
    delete[] h_dec_result;
}